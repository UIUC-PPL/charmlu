#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "stdio.h"
#include "gpu_kernel.h"
#include "assert.h"
#include "c_common.h"

char buf[100000];

__global__ void GPUKernel(float *Lm, float *Um, float *LUm,
                          int *Lstart, int *Lend,
                          int *Ustart, int *Uend,
                          int block, int total) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int lstart = Lstart[i], ustart = Ustart[i];
  float val = 0.0;
  int offsetx = (i % (block * block)) % block;
  int offsety = (i % (block * block)) / block;

  for (int k = 0; k < block; k++) {
    int i1 = (k + block * offsety) + lstart;
    int i2 = (k * block + offsetx) + ustart;

    float elm1 = Lm[i1];
    float elm2 = Um[i2];
    val += elm1 * elm2;
  }

  LUm[i] += -1 * val;
}

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }            
}

void checkCUDAKernelError(const char *msg) {
  hipError_t err = hipDeviceSynchronize();
  if( hipSuccess != err) {
    sprintf(buf, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }                        
}

static int n = MAX_OFFLOAD_SIZE;
static float *d_Lm;
static float *d_Um;
static float *d_LUm;
static int *d_Lstart;
static int *d_Lend;
static int *d_Ustart;
static int *d_Uend;
static int allocated = 0;
  
void GPUallocate() {
  if (!allocated) {
    printf("allocation happening\n");
    hipMalloc((void **) &d_Lm, n * sizeof(float));
    hipMalloc((void **) &d_Um, n * sizeof(float));
    hipMalloc((void **) &d_LUm, n * sizeof(float));
    hipMalloc((void **) &d_Lstart, n * sizeof(int));
    hipMalloc((void **) &d_Lend, n * sizeof(int));
    hipMalloc((void **) &d_Ustart, n * sizeof(int));
    hipMalloc((void **) &d_Uend, n * sizeof(int));
    checkCUDAError("mem allocation");
    allocated = 1;
    printf("allocation finished\n");
  }
}

void GPUKernelDGEMM(float Lm[], float Um[], float LUm[], 
                    int Lstart[], int Lend[], int Ustart[], int Uend[],
                    int block, int total) {
  /*for (int i = 0; i < sn; i++) {
    assert(sStart[i] >= 0);
    assert(sStart[i] <= sEnd[i]);
    assert(sEnd[i] < fn);
    assert(sEnd[i] - sStart[i] > 0 && sEnd[i] - sStart[i] < 5000);
  }

  for (int i = 0; i < fn; i++) {
    assert(fStart[i] >= 0);
    assert(fStart[i] <= fEnd[i]);
    assert(fEnd[i] < sn);
    assert(fEnd[i] - fStart[i] > 0 && fEnd[i] - fStart[i] < 5000);
    }*/
	
  /*int findex = fn - 1;
    int sindex = sn - 1;*/

  //printf("fStart[0] = %d, fEnd[0] = %d, fStart[%d] = %d, fEnd[%d] = %d\n", fStart[0], fEnd[0], findex, fStart[findex], findex, fEnd[findex]);
  //printf("sStart[0] = %d, sEnd[0] = %d, sStart[%d] = %d, sEnd[%d] = %d\n", sStart[0], sEnd[0], sindex, sStart[sindex], sindex, sEnd[sindex]);

  //printf("total = %d, block = %d\n", total, block);

  size_t dsize = total * sizeof(float);
  size_t isize = total * sizeof(int);

  //printf("dsize = %d, isize = %d\n", (int)dsize, (int)isize);
	
  /*assert(sizes < (n * sizeof(float)) );
    assert(sizef < (n * sizeof(float)) );
    assert(sizesi < (n * sizeof(int)) );
    assert(sizefi < (n * sizeof(int)) );*/

  GPUallocate();

  hipMemcpy(d_Lm, Lm, dsize, hipMemcpyHostToDevice);
  hipMemcpy(d_Um, Um, dsize, hipMemcpyHostToDevice);
  hipMemcpy(d_LUm, LUm, dsize, hipMemcpyHostToDevice);
  hipMemcpy(d_Lstart, Lstart, isize, hipMemcpyHostToDevice);
  hipMemcpy(d_Lend, Lend, isize, hipMemcpyHostToDevice);
  hipMemcpy(d_Ustart, Ustart, isize, hipMemcpyHostToDevice);
  hipMemcpy(d_Uend, Uend, isize, hipMemcpyHostToDevice);

  checkCUDAError("mem copy to device");

  /*for (int i = 0; i < total; i++) {
    printf("before LUm[%d] = %f\n", i, LUm[i]);
  }

  for (int i = 0; i < total; i++) {
    printf("Lstart[%d] = %d\n", i, Lstart[i]);
  }

  for (int i = 0; i < total; i++) {
    printf("Ustart[%d] = %d\n", i, Ustart[i]);
  }

  for (int i = 0; i < total; i++) {
    printf("before Lm[%d] = %f\n", i, Lm[i]);
  }

  for (int i = 0; i < total; i++) {
    printf("before Um[%d] = %f\n", i, Um[i]);
  }

  printf("TOTAL = %d\n", total);*/

  int blockSize = 16;
  int nBlocks = total/blockSize + (total%blockSize == 0?0:1);

  GPUKernel<<<nBlocks, blockSize>>>(d_Lm, d_Um, d_LUm,
                                    d_Lstart, d_Lend,
                                    d_Ustart, d_Uend,
                                    block, total);
  checkCUDAKernelError("kernel execute");

  hipMemcpy(LUm, d_LUm, dsize, hipMemcpyDeviceToHost);

  /*for (int i = 0; i < total; i++) {
    printf("after LUm[%d] = %f\n", i, LUm[i]);
    }*/

  checkCUDAError("mem copy from device");
}
